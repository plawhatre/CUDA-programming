#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	int size = 4;

	const int a[] = {1, 2, 3, 4};
	const int b[size] = {100, 200, 300, 400};
	int c[size] = {0,0,0,0};

	int *da, *db, *dc;

	hipMalloc((void **) & da, size*sizeof(int));
	hipMalloc((void **) & db, size*sizeof(int));
	hipMalloc((void **) & dc, size*sizeof(int));

	hipMemcpy(da, a, size*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(db, b, size*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dc, c, size*sizeof(int),hipMemcpyHostToDevice);


	add <<<1, size>>>(dc, da, db);

	hipMemcpy(c, dc, size*sizeof(int), hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	for (int x=0; x<size; x++)
	{
		printf("%d\n", c[x]);
	}


	return 0;


}