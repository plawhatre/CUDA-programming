
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuError(ans) {gpuChk((ans), __FILE__, __LINE__);}
inline void gpuChk(hipError_t error, const char *file, int line, bool abort=true)
{
	if (error != hipSuccess)
	{
		fprintf(stderr, "Error is: %s in file %s at line %d\n", 
			hipGetErrorString(error), file, line);
		if (abort==true){exit(error);}
	}
	else
	{
		printf("\nError free!\n");
	}
}

__global__ void msg(int *a)
{
	int i = threadIdx.x;
	printf("index: %d, value: %d\n", i, a[i]);
}

int main()
{
	int *da;

	int a[] = {11,22,33,44};
	
	hipError_t error;
	error = hipMalloc(&da, 4*sizeof(int));


	if (error != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(error));
	}
	else
	{
		printf("cool!");
	}


	gpuError(hipMalloc(&da, 4*sizeof(int)));
	hipDeviceSynchronize();
	return 0;
}