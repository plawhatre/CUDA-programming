
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

int seq_reduction(int *a, int size)
{
	int sum = 0;

	for (int i=0; i<size; i++)
	{
		sum += a[i];
	}
	return sum;
}

__global__ void interleaved_pair(int *a, int *tmp, int size)
{
	int tid = threadIdx.x;
	int gid = tid + blockIdx.x * blockDim.x;


	if (gid > size)
		return;

	for (int offset=blockDim.x/2; offset > 0; offset/=2 )
	{
		if (tid < offset)
		{
			a[gid] += a[gid+offset];
		}
		__syncthreads();
	}

	if (tid==0)
	{
		tmp[blockIdx.x] = a[gid];
	}

}

__global__ void alternative_interleaved_pair(int *a, int *tmp, int size)
{
	// same as previous implementation
	int tid = threadIdx.x;
	int gid = tid + blockIdx.x * blockDim.x;
	int *a_new = a + blockIdx.x * blockDim.x;

	if (gid > size)
		return;

	for (int offset=blockDim.x/2; offset > 0; offset/=2 )
	{
		if (tid < offset)
		{
			a_new[tid] += a_new[tid+offset];
		}
		__syncthreads();
	}

	if (tid==0)
	{
		tmp[blockIdx.x] = a_new[tid];
	}

}


int main()
{
	int size=1024*50;
	int a[size];
	// for (int i=0; i <size; i++){a[i] = (rand()%10);}
	for (int i=0; i <size; i++){a[i] = 1;}
	
	// CPU implementtion
	clock_t start_cpu, end_cpu;
	double cpu_time;

	start_cpu = clock();
	int res = seq_reduction(a, size);
	end_cpu = clock();

	cpu_time = end_cpu - start_cpu;

	printf("CPU result %d\ntime taken %f\n", res, cpu_time);

	// GPU implementation

	int *da, *dtmp;

	hipMalloc((void**)&da, size*sizeof(int));
	hipMemcpy(da, a, size*sizeof(int), hipMemcpyHostToDevice);

	dim3 block(128);
	dim3 grid(size/block.x);

	hipMalloc((void**)&dtmp, grid.x*sizeof(int));
	hipMemset(dtmp, 0, grid.x*sizeof(int));

	clock_t start_gpu, end_gpu;
	double gpu_time;

	start_gpu = clock();

	interleaved_pair<<<grid, block>>>(da, dtmp, size);

	hipMemcpy(a, da, size*sizeof(int), hipMemcpyDeviceToHost);
	int tmp[grid.x];
	hipMemcpy(tmp, dtmp, grid.x*sizeof(int), hipMemcpyDeviceToHost);

	int sm = seq_reduction(tmp, grid.x);
	end_gpu = clock();

	gpu_time = end_gpu - start_gpu;

	printf("\n\nGPU result %d\ntime taken %f\n", sm, gpu_time);

	hipDeviceSynchronize();

	return 0;
}