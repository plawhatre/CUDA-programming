
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main()
{
    int c ,* devc ;
    hipMalloc((void **) & devc , sizeof ( int ) ) ;
    
    add <<<1 ,1 >>>(2 ,5 , devc ) ;
    hipMemcpy (& c , devc , sizeof ( int ) , hipMemcpyDeviceToHost) ;
    
    hipFree(devc) ;
    
    printf("2+5 = %d \n" , c ) ;
    return 0;
}