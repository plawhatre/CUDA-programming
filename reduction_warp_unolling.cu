
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

int seq_reduction(int *a, int size)
{
	int sum = 0;

	for (int i=0; i<size; i++)
	{
		sum += a[i];
	}
	return sum;
}

__global__ void warp_unrolling(int *a, int *tmp, int size)
{
	int tid = threadIdx.x;
	int gid = tid + blockIdx.x * blockDim.x;
	int *a_new = a + blockIdx.x * blockDim.x;

	if (gid > size)
		return;

	for (int offset=blockDim.x/2; offset >= 64; offset/=2 )
	{
		if (tid < offset)
		{
			a_new[tid] += a_new[tid+offset];
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		volatile int *v = a_new;
		v[tid] += v[tid+32];
		v[tid] += v[tid+16];
		v[tid] += v[tid+8];
		v[tid] += v[tid+4];
		v[tid] += v[tid+2];
		v[tid] += v[tid+1];
	}

	if (tid==0)
	{
		tmp[blockIdx.x] = a_new[tid];
	}

}



int main()
{
	int size=1024*50;
	int a[size];
	// for (int i=0; i <size; i++){a[i] = (rand()%10);}
	for (int i=0; i <size; i++){a[i] = 1;}
	
	// CPU implementtion
	clock_t start_cpu, end_cpu;
	double cpu_time;

	start_cpu = clock();
	int res = seq_reduction(a, size);
	end_cpu = clock();

	cpu_time = end_cpu - start_cpu;

	printf("CPU result %d\ntime taken %f\n", res, cpu_time);

	// GPU implementation

	int *da, *dtmp;

	hipMalloc((void**)&da, size*sizeof(int));
	hipMemcpy(da, a, size*sizeof(int), hipMemcpyHostToDevice);

	dim3 block(128);
	dim3 grid(size/block.x);

	hipMalloc((void**)&dtmp, grid.x*sizeof(int));
	hipMemset(dtmp, 0, grid.x*sizeof(int));

	clock_t start_gpu, end_gpu;
	double gpu_time;

	start_gpu = clock();

	warp_unrolling<<<grid, block>>>(da, dtmp, size);

	hipMemcpy(a, da, size*sizeof(int), hipMemcpyDeviceToHost);
	int tmp[grid.x];
	hipMemcpy(tmp, dtmp, grid.x*sizeof(int), hipMemcpyDeviceToHost);

	int sm = seq_reduction(tmp, grid.x);
	end_gpu = clock();

	gpu_time = end_gpu - start_gpu;

	printf("\n\nGPU result %d\ntime taken %f\n", sm, gpu_time);

	hipDeviceSynchronize();

	return 0;
}