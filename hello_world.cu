#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello(void)
{
	printf("hello cuda \n");
}

int main()
{
	dim3 block(4,2);
	dim3 grid(8/block.x,16/block.y);

	hello<<<grid, block>>>();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}